
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

const int MAXX = 1e8;


#define CSC(call)                            \
do {                                \
  hipError_t res = call;                      \
  if (res != hipSuccess) {                    \
    fprintf(stderr, "ERROR in %s:%d. Message: %s\n",      \
        __FILE__, __LINE__, hipGetErrorString(res));    \
    exit(0);                          \
  }                                \
} while(0)


struct pnt {
    int x, y;
};


//__global__ void kernel(uchar4 *out, int w, int h, int nc) {
//    int idx = blockDim.x * blockIdx.x + threadIdx.x;
//    int idy = blockDim.y * blockIdx.y + threadIdx.y;
//    int offsetx = blockDim.x * gridDim.x;
//    int offsety = blockDim.y * gridDim.y;
//    int x, y;
//
//
//
//    for(y = idy; y < h; y += offsety) {
//        for(x = idx; x < w; x += offsetx) {
//
//
//        }
//    }
//}

int main() {


    int w, h;
    char inputFile[256], outputFile[256];
    cin >> inputFile >> outputFile;

    FILE *fp = fopen(inputFile, "rb");
    fread(&w, sizeof(int), 1, fp);
    fread(&h, sizeof(int), 1, fp);

    uchar4 *data = (uchar4 *) malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    int nc, np;
    cin >> nc;
    vector<vector<pnt>> classes(nc);
    int4 avg[32];
    double cov[32][3][3];
    double cov_inv[32][3][3];
    double dets[32];
    for (int i = 0; i < nc; ++i) {
        cin >> np;
        classes[i].resize(np);
            // input + counting averages
        for (int j = 0; j < np; ++j) {
            cin >> classes[i][j].x >> classes[i][j].y;
            uchar4 ps = data[classes[i][j].y * w + classes[i][j].x];
            avg[i].x += ps.x;
            avg[i].y += ps.y;
            avg[i].z += ps.z;
        }
        avg[i].x /= np;
        avg[i].y /= np;
        avg[i].z /= np;

        // counting cov
        for (int j = 0; j < np; ++j) {
            uchar4 ps = data[classes[i][j].y * w + classes[i][j].x];


            int diff[3];
            diff[0] = ps.x - avg[i].x;
            diff[1] = ps.y - avg[i].y;
            diff[2] = ps.z - avg[i].z;

            for (int k = 0; k < 3; ++k) {
                for (int m = 0; m < 3; ++m) {
                    cov[i][k][m] += diff[k] * diff[m];
                }
            }
        }
        for (int k = 0; k < 3; ++k) {
            for (int m = 0; m < 3; ++m) {
                cov[i][k][m] /= (np - 1);
            }
        }

        // counting cov_inverse + determinants
        double det = cov[i][0][0] * (cov[i][1][1] * cov[i][2][2] - cov[i][2][1] * cov[i][1][2])
                     - cov[i][0][1] * (cov[i][1][0] * cov[i][2][2] - cov[i][2][0] * cov[i][1][2])
                     + cov[i][0][2] * (cov[i][1][0] * cov[i][2][1] - cov[i][2][0] * cov[i][1][1]);


        cov_inv[i][0][0] = (cov[i][1][1] * cov[i][2][2] - cov[i][2][1] * cov[i][1][2]) / det;
        cov_inv[i][1][0] = -(cov[i][1][0] * cov[i][2][2] - cov[i][2][0] * cov[i][1][2]) / det;
        cov_inv[i][2][0] = (cov[i][1][0] * cov[i][2][1] - cov[i][2][0] * cov[i][1][1]) / det;

        cov_inv[i][0][1] = -(cov[i][0][1] * cov[i][2][2] - cov[i][2][1] * cov[i][0][2]) / det;
        cov_inv[i][1][1] = (cov[i][0][0] * cov[i][2][2] - cov[i][2][0] * cov[i][0][2]) / det;
        cov_inv[i][2][1] = -(cov[i][0][0] * cov[i][2][1] - cov[i][2][0] * cov[i][0][1]) / det;

        cov_inv[i][0][2] = (cov[i][0][1] * cov[i][1][2] - cov[i][1][1] * cov[i][0][2]) / det;
        cov_inv[i][1][2] = -(cov[i][0][0] * cov[i][1][2] - cov[i][1][0] * cov[i][0][2]) / det;
        cov_inv[i][2][2] = (cov[i][0][0] * cov[i][1][1] - cov[i][1][0] * cov[i][0][1]) / det;

        dets[i] = det;
    }



    for (int y = 0; y < h; ++y){
        for (int x = 0; x < w; ++x){
            uchar4 ps = data[y * w + x];

            double mx = -MAXX;
            int idx = -1;
            for (int i = 0; i < nc; ++i){

                int diff[3];
                diff[0] = ps.x - avg[i].x;
                diff[1] = ps.y - avg[i].y;
                diff[2] = ps.z - avg[i].z;

                double tmp[3];
                for(int j = 0; j < 3; ++j){
                    tmp[j] = 0;
                    for(int k = 0; k < 3; ++k){
                        tmp[j] += (diff[k] * cov_inv[i][k][j]);
                    }
                }
                double ans = 0;
                for(int j = 0; j < 3; ++j){
                    ans += (tmp[j] * diff[j]);
                }
                ans = -ans - log(abs(dets[i]));

                if(ans > mx){
                    mx = ans;
                    idx = i;
                }
            }

            data[y * w + x].w = idx;
        }
    }



    fp = fopen(outputFile, "wb");
    fwrite(&w, sizeof(int), 1, fp);
    fwrite(&h, sizeof(int), 1, fp);
    fwrite(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    free(data);
    return 0;
}
