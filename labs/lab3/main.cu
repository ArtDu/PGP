
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)



int main() {


	int w, h;
	char inputFile[256], outputFile[256];
	scanf("%s %s %d %d", inputFile, outputFile);

	FILE *fp = fopen(inputFile, "rb");
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);

	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);


	fp = fopen(outputFile, "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
	return 0;
}
