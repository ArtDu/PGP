#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include <unistd.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;
using namespace thrust;

#define CSC(call)                           \
do {                                \
  hipError_t res = call;                     \
  if (res != hipSuccess) {                   \
    fprintf(stderr, "ERROR in %s:%d. Message: %s\n",      \
        __FILE__, __LINE__, hipGetErrorString(res));   \
    exit(0);                          \
  }                               \
} while(0)

#define BLOCKS 1
#define BUCKET_SIZE 1024

__global__ void bitonic_sort_step(int *values, int sz, int odd) {
    __shared__ int shared[BUCKET_SIZE];

    int *dev_values, k = BUCKET_SIZE;
    unsigned int i, j, ixj, temp; /* Sorting partners: i and ixj */
    int block_id = blockIdx.x;
    int block_offset = gridDim.x;
    i = threadIdx.x;

    if(odd)
        for (int step = (BUCKET_SIZE / 2) + block_id * BUCKET_SIZE; step + BUCKET_SIZE < sz; step = step + block_offset * BUCKET_SIZE) {
            dev_values = values + step;

            if (i >= k / 2) {
                int diff = i - k / 2;
                shared[i] = dev_values[k - 1 - diff];
            } else {
                shared[i] = dev_values[i];
            }

            __syncthreads();

            for (j = k >> 1; j > 0; j = j >> 1) {
                ixj = i ^ j;
                /* The threads with the lowest ids sort the array. */
                if ((ixj) > i) {
                    if ((i & k) == 0) {
                        /* Sort ascending */
                        if (shared[i] > shared[ixj]) {
                            /* exchange(i,ixj); */
                            temp = shared[i];
                            shared[i] = shared[ixj];
                            shared[ixj] = temp;
                        }
                    }
                    if ((i & k) != 0) {
                        /* Sort descending */
                        if (shared[i] < shared[ixj]) {
                            /* exchange(i,ixj); */
                            temp = shared[i];
                            shared[i] = shared[ixj];
                            shared[ixj] = temp;
                        }
                    }
                }
                __syncthreads();
            }
            dev_values[i] = shared[i];
        }
    else
        for (int step = block_id * BUCKET_SIZE; step < sz; step = step + block_offset * BUCKET_SIZE) {
            dev_values = values + step;

            if (i >= k / 2) {
                int diff = i - k / 2;
                shared[i] = dev_values[k - 1 - diff];
            } else {
                shared[i] = dev_values[i];
            }

            __syncthreads();

            for (j = k >> 1; j > 0; j = j >> 1) {
                ixj = i ^ j;
                /* The threads with the lowest ids sort the array. */
                if ((ixj) > i) {
                    if ((i & k) == 0) {
                        /* Sort ascending */
                        if (shared[i] > shared[ixj]) {
                            /* exchange(i,ixj); */
                            temp = shared[i];
                            shared[i] = shared[ixj];
                            shared[ixj] = temp;
                        }
                    }
                    if ((i & k) != 0) {
                        /* Sort descending */
                        if (shared[i] < shared[ixj]) {
                            /* exchange(i,ixj); */
                            temp = shared[i];
                            shared[i] = shared[ixj];
                            shared[ixj] = temp;
                        }
                    }
                }
                __syncthreads();
            }
            dev_values[i] = shared[i];
        }

}

void bitonic_sort(int *values, int sz) {
    for (int _i = 0; _i < 2 * (sz / BUCKET_SIZE); ++_i) {
        bitonic_sort_step<<<BLOCKS, BUCKET_SIZE>>>(values, sz, _i % 2);
    }
}


__global__ void oddeven_sort(int *dev_values, int sz) {
    __shared__ int shared[BUCKET_SIZE];

    int id = threadIdx.x;
    int block_id = blockIdx.x;
    int block_offset = gridDim.x;
    int odd, i, n = BUCKET_SIZE;
    int *values;

    for (int j = block_id * BUCKET_SIZE; j < sz; j = j + block_offset * BUCKET_SIZE) {
        values = dev_values + j;
        shared[id] = values[id];
        __syncthreads();

        for (i = 0; i < n; i++) {
            odd = i % 2;
            if (odd == 0 && id % 2 == 0 && id + 1 < n) {
                if (shared[id] > shared[id + 1]) {
                    int tmp = shared[id];
                    shared[id] = shared[id + 1];
                    shared[id + 1] = tmp;
                }
            }
            if (odd == 1 && id % 2 == 1 && id + 1 < n) {
                if (shared[id] > shared[id + 1]) {
                    int tmp = shared[id];
                    shared[id] = shared[id + 1];
                    shared[id + 1] = tmp;
                }
            }

            __syncthreads();
        }
        values[id] = shared[id];
    }


}


void parallel_sort(int *values, int new_sz) {
    int *dev_values;
    size_t size = new_sz * sizeof(int);

    CSC(hipMalloc(&dev_values, size));
    CSC(hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice));

    // sort all buckets by odd-even sort
    oddeven_sort<<<BLOCKS, BUCKET_SIZE>>>(dev_values, new_sz);

    // sort buckets between themselves by odd-even merge sort
//    bitonic_sort<<<BLOCKS, BUCKET_SIZE>>>(dev_values, new_sz);
    bitonic_sort(dev_values, new_sz);


    CSC(hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost));
//    CSC(hipFree(dev_values));
}

int main(int argc, char *argv[]) {
    ios_base::sync_with_stdio(false);
    cin.tie(NULL);
    cout.tie(NULL);



    int sz;
//    scanf("%d", &sz);
    fread(&sz, sizeof(int), 1, stdin);
//    fprintf(stderr, "%d\n", sz);

    int new_sz = ceil((double)sz / BUCKET_SIZE) * BUCKET_SIZE;
//    fprintf(stderr, "%d\n", new_sz);

    int *values = (int*) malloc( new_sz * sizeof(int));
    fread(values, sizeof(int), sz, stdin);
    for (int i = sz; i < new_sz; ++i){
        values[i] = INT_MAX;
    }


//    int i;
//    for (i = 0; i < sz; ++i)
//        scanf("%d", &values[i]);
//    for(; i < new_sz; ++i)
//        values[i] = INT_MAX;

    hipEvent_t start, stop;
    float gpu_time = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    parallel_sort(values, new_sz);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    cerr << "time:\n";
    cerr << "blocks = " << BLOCKS << "; threads = " << BUCKET_SIZE << "\n";
    cerr << gpu_time << endl;

    fwrite(values, sizeof(int), sz, stdout);
//    for (int i = 0; i < sz; i++) {
//        fwrite(&values[i], sizeof(int), 1, stdout);
//        fprintf(stderr, "%d ", values[i]);
//    }
//    fprintf(stderr, "\n");
//    free(values);
    return 0;
}

